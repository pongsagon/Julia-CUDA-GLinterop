#include "hip/hip_runtime.h"

//  interop with opengl, glfw
//  use win32 to handle touch input
//  use stb_image for image loading
//  load img, convert to grayscale using cuda, display using opengl




#define GLFW_EXPOSE_NATIVE_WIN32
#define GLFW_EXPOSE_NATIVE_WGL
#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))


#define STB_IMAGE_IMPLEMENTATION
#include "includes/stb_image.h"
#include "includes/KHR/khrplatform.h"
#include "includes/glew.h"
#include "includes/GLFW/glfw3.h"
#include "includes/GLFW/glfw3native.h"
#include "includes/shader.h"

// must include after gl lib
#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include <windows.h>
#include <sstream>
#include <iostream>
#include <time.h>
#include <float.h>

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 800;
double prevTime = 0.0;
double currTime = 0.0;

// Windows
HWND handle;
WNDPROC currentWndProc;
MSG Msg;
WNDPROC btnWndProc;
std::stringstream ss;

// Touch
#define MAXPOINTS 20
// You will use this array to track touch points
int points[MAXPOINTS][2];
// You will use this array to switch the color / track ids
int idLookup[MAXPOINTS];
int last_points[MAXPOINTS][2];
int diff_points[MAXPOINTS][2];


// cuda opengl interop
GLuint shDrawTex;           // shader
GLuint tex_cudaResult;      // result texture to glBindTexture(GL_TEXTURE_2D, texture);
unsigned int* cuda_dest_resource;  // output from cuda
struct hipGraphicsResource* cuda_tex_result_resource;

// fractal
#define WIDTH 1900
#define HEIGHT 1000
//double cx = -0.162;
//double cy = 1.04;
////
//double cx = 0.3;
//double cy = -0.01;
//
//double cx = -1.476;
//double cy = 0.0;
////
double cx = -0.79;
double cy = 0.15;
////
//double cx = -0.12;
//double cy = -0.77;
//
//double cx = 0.28;
//double cy = 0.008;
double scale = 1.5;
double panX = 0.0f;
double panY = 0.0f;

// ---------------------------------------
// Touch handler
// ---------------------------------------

// This function is used to return an index given an ID
int GetContactIndex(int dwID) {
    for (int i = 0; i < MAXPOINTS; i++) {
        if (idLookup[i] == dwID) {
            return i;
        }
    }

    for (int i = 0; i < MAXPOINTS; i++) {
        if (idLookup[i] == -1) {
            idLookup[i] = dwID;
            return i;
        }
    }
    // Out of contacts
    return -1;
}

// Mark the specified index as initialized for new use
BOOL RemoveContactIndex(int index) {
    if (index >= 0 && index < MAXPOINTS) {
        idLookup[index] = -1;
        return true;
    }

    return false;
}

LRESULT OnTouch(HWND hWnd, WPARAM wParam, LPARAM lParam) {
    BOOL bHandled = FALSE;
    UINT cInputs = LOWORD(wParam);
    PTOUCHINPUT pInputs = new TOUCHINPUT[cInputs];
    POINT ptInput;
    if (pInputs) {
if (GetTouchInputInfo((HTOUCHINPUT)lParam, cInputs, pInputs, sizeof(TOUCHINPUT))) {
    for (UINT i = 0; i < cInputs; i++) {
        TOUCHINPUT ti = pInputs[i];
        int index = GetContactIndex(ti.dwID);
        if (ti.dwID != 0 && index < MAXPOINTS) {

            // Do something with your touch input handle
            ptInput.x = TOUCH_COORD_TO_PIXEL(ti.x);
            ptInput.y = TOUCH_COORD_TO_PIXEL(ti.y);
            ScreenToClient(hWnd, &ptInput);

            if (ti.dwFlags & TOUCHEVENTF_UP) {
                points[index][0] = -1;
                points[index][1] = -1;
                last_points[index][0] = -1;
                last_points[index][1] = -1;
                diff_points[index][0] = 0;
                diff_points[index][1] = 0;

                // Remove the old contact index to make it available for the new incremented dwID.
                // On some touch devices, the dwID value is continuously incremented.
                RemoveContactIndex(index);
            }
            else {
                if (points[index][0] > 0) {
                    last_points[index][0] = points[index][0];
                    last_points[index][1] = points[index][1];
                }

                points[index][0] = ptInput.x;
                points[index][1] = ptInput.y;

                if (last_points[index][0] > 0) {
                    diff_points[index][0] = points[index][0] - last_points[index][0];
                    diff_points[index][1] = points[index][1] - last_points[index][1];
                }
            }
        }
    }
    bHandled = TRUE;
}
else {
    /* handle the error here */
}
delete[] pInputs;
    }
    else {
    /* handle the error here, probably out of memory */
    }
    if (bHandled) {
        // if you handled the message, close the touch input handle and return
        CloseTouchInputHandle((HTOUCHINPUT)lParam);
        return 0;
    }
    else {
        // if you didn't handle the message, let DefWindowProc handle it
        return DefWindowProc(hWnd, WM_TOUCH, wParam, lParam);
    }
}

LRESULT CALLBACK SubclassWindowProc(HWND hWnd, UINT uMsg, WPARAM wParam, LPARAM lParam) {
    switch (uMsg) {
    case WM_TOUCH:
        OnTouch(hWnd, wParam, lParam);
        break;
    case WM_LBUTTONDOWN:
    {

    }
    break;
    case WM_CLOSE:
        DestroyWindow(hWnd);
        break;
    case WM_DESTROY:
        PostQuitMessage(0);
        break;
    }

    return CallWindowProc(btnWndProc, hWnd, uMsg, wParam, lParam);
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        OutputDebugStringA("Key W press \n");



    // 3 point, change to Mandlebrot
    if (points[2][0] >= 0) {
        cx = cx + 0.0001 * scale * diff_points[0][0];
        cy = cy + 0.0001 * scale * diff_points[0][1];
    }
    // 2 point, zoom
    else if (points[1][0] >= 0) {
        if (diff_points[0][1] > 0.01f){
            scale *= 1.1;
        }
        else if (diff_points[0][1] < -0.01f) {
            scale *= 0.9;
        }
    }
    // 1 point, pan
    else if (points[0][0] >= 0) {
        panX = panX + 0.0011 * scale * diff_points[0][0];
        panY = panY + 0.0011 * scale * diff_points[0][1];
    }


}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and 
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}





// ---------------------------------------
// CUDA code
// ---------------------------------------

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        ss << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        OutputDebugStringA(ss.str().c_str());
        hipDeviceReset();
        exit(99);
    }
}

__device__ int clamp(int x, int a, int b) { return MAX(a, MIN(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);

    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__
void img_process(unsigned char* in, unsigned int* out, int width, int height) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < 0 || row >= height || col < 0 || col >= width) return;

    int grey_offset = row * width + col;
    int rgb_offset = grey_offset * 3;

    unsigned char r = in[rgb_offset + 0];
    unsigned char g = in[rgb_offset + 1];
    unsigned char b = in[rgb_offset + 2];

    float gray = (0.21f * r + 0.71f * g + 0.07f * b);

    out[row * width + col] = rgbToInt(r,g,b);

}

struct hipComplex {
    double r;
    double i;
    __device__ hipComplex(double a, double b) : r(a), i(b) {}
    __device__ float magnitude2(void) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y, int width, int height, double cx, double cy, double scale, double panX, double panY) {
    //const float scale = 1.5;
    double jx = panX + scale * ((float)(width / 2 - x) / (width / 2));
    double jy = panY + scale * ((float)(height / 2 - y) / (width / 2));
    //hipComplex c(-0.8, 0.156);
    hipComplex c(cx, cy);
    //hipComplex c(jx, jy);		// for Mandelbrot
    hipComplex a(jx, jy);
    int i = 0;
    for (i = 0; i < 512; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return i;
    }
    return 0;
}


// 1 block 64 thread, 
__global__ void kernel(unsigned int* out, int width, int height, double cx, double cy, double scale, double panX, double panY) {
    // map from threadIdx/BlockIdx to pixel position
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < 0 || row >= height || col < 0 || col >= width) return;

    int offset = row * width + col;
    // now calculate the value at that position
    float juliaValue = julia(col, row, width, height, cx, cy, scale, panX, panY);
    juliaValue = juliaValue / 511.0f;

    unsigned char r = 255 * sqrtf(juliaValue);
    unsigned char g = 255 * powf(juliaValue,3);
    unsigned char b = 0;
    float bf = sinf(2 * 3.14159 * juliaValue);
    if (bf < 0) {
        b = 0;
    }
    else {
        b = 255 * bf;
    }

    out[row * width + col] = rgbToInt(r, g, b);

}


int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance,
    LPSTR lpCmdLine, int nCmdShow)
{
    // not need
    //cudaGLSetGLDevice(0);

    // glfw: initialize and configure
    // ------------------------------
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);


    // glfw window creation
    // --------------------
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "LearnOpenGL", NULL, NULL);

    handle = glfwGetWin32Window(window);
    btnWndProc = (WNDPROC)SetWindowLongPtrW(handle, GWLP_WNDPROC, (LONG_PTR)SubclassWindowProc);
    int touch_success = RegisterTouchWindow(handle, 0);

    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    // set this to 0, will swap at fullspeed, but app will close very slow, sometime hang
    glfwSwapInterval(1);

    // Initialize GLEW
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "Failed to initialize GLEW\n");
        getchar();
        glfwTerminate();
        return -1;
    }

    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);


    // init touch data
    for (int i = 0; i < MAXPOINTS; i++) {
        points[i][0] = -1;
        points[i][1] = -1;
        last_points[i][0] = -1;
        last_points[i][1] = -1;
        diff_points[i][0] = 0;
        diff_points[i][1] = 0;
        idLookup[i] = -1;
    }

    // gl init
    // ---------------------------------------
    Shader ourShader("tex.vs", "tex.fs");
    // set up vertex data (and buffer(s)) and configure vertex attributes
    // ------------------------------------------------------------------
    float vertices[] = {
        // positions          // texture coords
         1.0f,  1.0f, 0.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   0.0f, 1.0f  // top left 
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);
    // color attribute
    glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);


    // cuda mem out bind to tex
    // ---------------------------------------
    int num_texels = WIDTH * HEIGHT;
    int num_values = num_texels * 4;
    int size_tex_data = sizeof(GLubyte) * num_values;
    checkCudaErrors(hipMalloc((void**)&cuda_dest_resource, size_tex_data));

    // create a texture, output from cuda
    glGenTextures(1, &tex_cudaResult);
    glBindTexture(GL_TEXTURE_2D, tex_cudaResult);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_tex_result_resource, tex_cudaResult, GL_TEXTURE_2D,cudaGraphicsMapFlagsWriteDiscard));

    // fps
    prevTime = glfwGetTime();
    

    while (!glfwWindowShouldClose(window))//(Msg.message != WM_QUIT)
    {
        // fps
        /*currTime = glfwGetTime();
        double result = currTime - prevTime;
        ss << 1.0f/result << "\n";
        OutputDebugStringA(ss.str().c_str());
        ss.str("");*/

        processInput(window);


        glClearColor(0.3f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // begin measure gpu
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);


        // process img
        int num_thread = 16;
        dim3 dimBlock(num_thread, num_thread, 1);
        dim3 dimGrid(ceil((float)WIDTH / num_thread), ceil((float)HEIGHT / num_thread), 1);
        kernel<<<dimGrid, dimBlock >>> (cuda_dest_resource, WIDTH, HEIGHT, cx, cy, scale, panX, panY);

        //hipDeviceSynchronize();


        // copy cuda_dest_resource data to the texture
        hipArray* texture_ptr;
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_result_resource, 0));
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray( &texture_ptr, cuda_tex_result_resource, 0, 0));


        checkCudaErrors(hipMemcpyToArray(texture_ptr, 0, 0, cuda_dest_resource, size_tex_data, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_result_resource, 0));

        
        // end measure gpu
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        ss << elapsedTime << "ms\n";
        OutputDebugStringA(ss.str().c_str());
        ss.str("");
        hipEventDestroy(start);
        hipEventDestroy(stop);
        



        // render gl
        glUniform1i(glGetUniformLocation(ourShader.ID, "texture1"), 0);
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, tex_cudaResult);
        ourShader.use();
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
        


        // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
        // -------------------------------------------------------------------------------
        glfwSwapBuffers(window);
        glfwPollEvents();

        // fps
        prevTime = currTime;
    }


    // Free the device memory
    hipFree(cuda_dest_resource);


    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    

    // glfw: terminate, clearing all previously allocated GLFW resources.
    // ------------------------------------------------------------------
    glfwTerminate();
    return 0;
}

